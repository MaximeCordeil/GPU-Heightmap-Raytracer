#pragma once

#include <glm/glm.hpp>
#include <cstdio>

#include <hip/hip_runtime.h>

/*
* Code snippet from
* http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
*/
#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

namespace CudaSpace
{
	__device__ void rayTrace()
	{
	}
}
