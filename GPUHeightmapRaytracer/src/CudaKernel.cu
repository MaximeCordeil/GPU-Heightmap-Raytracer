#include "hip/hip_runtime.h"
﻿#include "CudaKernel.cuh"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <math_functions.hpp>

namespace CudaSpace
{
	__device__ bool use_color_map = false;
	__device__ float max_height = 0;
	__device__ float *point_buffer;
	__device__ int *LOD_indexes, *LOD_resolutions;
	__device__ CudaSpace::Color *color_map;
	__device__ int LOD_levels, stride_x = 1;
	__device__ glm::vec3 *frame_dimension;
	__device__ glm::vec3 *camera_forward;
	__device__ glm::vec3 *grid_camera_position;
	__device__ glm::ivec2 *texture_resolution;
	__device__ glm::ivec2 *point_buffer_resolution;
	__device__ glm::ivec2 *boundary;
	__device__ glm::mat3x3 *pixel_to_grid_matrix;

	/*
	* Get a colormap value from a height map index
	*/
	__device__ void getColorMapValue(int posX, int posZ, bool mirrorX, bool mirrorZ, Color& result)
	{
		if (mirrorX)
			posX = LOD_resolutions[0] - 1 - posX;
		if (mirrorZ)
			posZ = LOD_resolutions[0] - 1 - posZ;

		result = color_map[posX + posZ * LOD_resolutions[0]];
	}

	/*
	* Get a value based on max height
	*/
	__device__ void getHeightColorValue(float height, Color& result)
	{
		unsigned char r, g, b;
		height = height * 2 / max_height ;
		if(height > 1)
		{
			height -= 1;
			r = 255;
			g = 255 - height * 255;
			b = 0;
		}
		else
		{
			r = 255 * height;
			g = r;
			b = 255 - height * 255;
		}
		result = Color(r, g, b);
	}

	/*
	 * Retrieve the height value from point buffer based on LOD and position
	 */
	__device__ float getPointBufferValue(int posX, int posZ, bool mirrorX, bool mirrorZ, int LOD)
	{
		if (mirrorX)
			posX = LOD_resolutions[LOD] - 1 - posX;
		if (mirrorZ)
			posZ = LOD_resolutions[LOD] - 1 - posZ;

		return point_buffer[LOD_indexes[LOD] + posX + posZ * LOD_resolutions[LOD]];
	}

	/*
	 *Calculate exit point based on current ray position
	 */
	__device__ void calculateExitPointAndEdge(glm::vec3& entry, glm::vec3& direction, glm::vec3& exit, int &edge, int LOD)
	{
		float tX, tZ;
		tX = ((floor(entry.x / pow(2.f, LOD)) + 1) * pow(2.f, LOD) - entry.x) / direction.x;
		tZ = ((floor(entry.z / pow(2.f, LOD)) + 1) * pow(2.f, LOD) - entry.z) / direction.z;
		if(tX <= tZ)
		{
			exit = entry + tX * direction;
			exit.x = (floor(entry.x / pow(2.f, LOD)) + 1) * pow(2.f, LOD);
			edge = floor(exit.x / pow(2.f, LOD));
		}
		else
		{
			exit = entry + tZ * direction;
			exit.z = (floor(entry.z / pow(2.f, LOD)) + 1) * pow(2.f, LOD);
			edge = floor(exit.z / pow(2.f, LOD));
		}
	}

	/*
	 * Test if the ray intersects with the height field
	 */
	__device__ bool testIntersection(glm::vec3 &entry, glm::vec3 &exit, glm::vec3 &direction, bool mirrorX, bool mirrorZ, int &LOD)
	{
		bool result;
		float height;

		height = getPointBufferValue(floor(entry.x / pow(2.f, LOD)), floor(entry.z / pow(2.f, LOD)), mirrorX, mirrorZ, LOD);
		if(direction.y >= 0)
		{
			result = entry.y <= height;
		}
		else
		{
			result = exit.y <= height;
			if (result)
				entry += glm::max(0.f, (height - entry.y) / direction.y) * direction;
		}

		return result;		
	}


	/*
	 *	Dick, C., et al. (2009). GPU ray-casting for scalable terrain rendering. Proceedings of EUROGRAPHICS, Citeseer.
	 *	ray_direction MUST be normalized
	 */
	__device__ void castRay(glm::vec3& ray_position, glm::vec3& ray_direction, Color& result)
	{
		bool mirrorX, mirrorZ;
		glm::vec3 ray_exit;
		int edge;
		int LOD = LOD_levels - 1;
		bool intersection;

		/*Mirror direction to simplify algorithm*/
		if(ray_direction.x < 0)
		{
			mirrorX = true;
			ray_direction.x = -ray_direction.x;
			ray_position.x = point_buffer_resolution->x * pow(2.f, LOD) - ray_position.x;
		}
		else
		{
			mirrorX = false;
		}

		if(ray_direction.z < 0)
		{
			mirrorZ = true;
			ray_direction.z = -ray_direction.z;
			ray_position.z = point_buffer_resolution->y * pow(2.f, LOD) - ray_position.z;
		}
		else
		{
			mirrorZ = false;
		}

		/*Advance ray until it is outside of the buffer*/
		while(ray_position.x < boundary->x && ray_position.z < boundary->y && !(ray_direction.y > 0 && ray_position.y > max_height))
		{
			calculateExitPointAndEdge(ray_position, ray_direction, ray_exit, edge, LOD);
			intersection = testIntersection(ray_position, ray_exit, ray_direction, mirrorX, mirrorZ, LOD);
			if(intersection)
			{
				if (LOD > 0)
					LOD--;
				else
				{
					if (use_color_map)
						getColorMapValue(floor(ray_position.x), floor(ray_position.z), mirrorX, mirrorZ, result);
					else
						getHeightColorValue(ray_position.y, result);
					return;
				}
				
			}
			else
			{
				LOD = glm::min(LOD + 1 - (edge % 2), LOD_levels - 1);
				ray_position = ray_exit;			
			}
		}
	}
	
	/*
	 * Converts a pixel position to the grid space
	 * Pinhole camera model - From: Realistic Ray Tracing by Peter Shirley, pages 37-42
	 */
	__device__ glm::vec3 viewToGridSpace(glm::ivec2 &pixel_position)
	{
		glm::vec3 result = glm::vec3(
			 frame_dimension->x / 2.0f - (frame_dimension->x) * pixel_position.x / (texture_resolution->x - 1),
			-frame_dimension->y / 2.0f + (frame_dimension->y) * pixel_position.y / (texture_resolution->y - 1),
			-frame_dimension->z);
		return result;
	}

	/*
	 * Start the ray tracing algorithm for each pixel
	 */
	__global__ void cuda_rayTrace(unsigned char* color_buffer)
	{
		/*2D Grid and Block*/
		int pixel_x, pixel_y, threadId;
		
		pixel_x = blockIdx.x * blockDim.x + threadIdx.x;
		pixel_y = blockIdx.y * blockDim.y + threadIdx.y;
		threadId = pixel_x + pixel_y * texture_resolution->x;
		
		Color color_value(static_cast<unsigned char>(200), static_cast<unsigned char>(200), static_cast<unsigned char>(200));
		glm::vec3 ray_direction, ray_position;
		glm::ivec2 pixel_position;

		/*Get the pixel position of this thread*/
		pixel_position = glm::ivec2(pixel_x, pixel_y);

		/*Calculate ray direction and cast ray*/
		ray_direction = *pixel_to_grid_matrix * viewToGridSpace(pixel_position);

		ray_position = ray_direction + *grid_camera_position;
		ray_direction = normalize(ray_direction);
		castRay(ray_position, ray_direction, color_value);
		
		//GL_RGB
		color_buffer[threadId * 3] = color_value.r;
		color_buffer[threadId * 3 + 1] = color_value.g;
		color_buffer[threadId * 3 + 2] = color_value.b;
	}

	/*
	* Set device parameters
	*/
	__global__ void cuda_setParameters(glm::vec3 frame_dim, glm::vec3 camera_for, glm::vec3 grid_camera_pos, bool use_color, float max_height)
	{
		*frame_dimension = frame_dim;
		*grid_camera_position = grid_camera_pos;
		use_color_map = use_color;
		CudaSpace::max_height = max_height;

		/*Basis change matrix from view to grid space*/
		glm::vec3 u, v, w;
		w = -camera_for;
		u = glm::normalize(glm::cross(glm::vec3(0, 100, 0), w));
		v = glm::cross(w, u);
		*pixel_to_grid_matrix = glm::mat3x3(u,v,w);
	}

	/* 
	 * Initialize device 
	 */
	__global__ void cuda_initializeDeviceVariables(glm::ivec2 point_buffer_resolution, glm::ivec2 texture_resolution, float* point_buffer, CudaSpace::Color *color_map, int LOD_levels, int stride_x, float max_height)
	{
		CudaSpace::texture_resolution = new glm::ivec2();
		CudaSpace::point_buffer_resolution = new glm::ivec2();

		LOD_indexes = new int[LOD_levels]();
		LOD_resolutions = new int[LOD_levels]();
		LOD_resolutions[LOD_levels - 1] = point_buffer_resolution.x;
		LOD_indexes[LOD_levels - 1] = 0;
		for(auto i = LOD_levels - 2; i >= 0; i--)
		{
			LOD_indexes[i] = LOD_indexes[i + 1] + LOD_resolutions[i + 1] * LOD_resolutions[i + 1];
			LOD_resolutions[i] = LOD_resolutions[i + 1] * 2;
		}

		boundary = new glm::ivec2(LOD_resolutions[0], LOD_resolutions[0]);
		frame_dimension = new glm::vec3();
		pixel_to_grid_matrix = new glm::mat3x3();
		grid_camera_position = new glm::vec3();

		*CudaSpace::point_buffer_resolution = point_buffer_resolution;
		*CudaSpace::texture_resolution = texture_resolution;
		CudaSpace::point_buffer = point_buffer;
		CudaSpace::color_map = color_map;
		CudaSpace::LOD_levels = LOD_levels;
		CudaSpace::stride_x = stride_x;
		CudaSpace::max_height = max_height;
	}

	/*
	* Free device's variables
	*/
	__global__ void cuda_freeDeviceVariables()
	{
		delete(grid_camera_position);
		delete(texture_resolution);
		delete(frame_dimension);
		delete(pixel_to_grid_matrix);
		delete(point_buffer_resolution);
		delete[](LOD_indexes);
		delete[](LOD_resolutions);
	}

	/*
	 * Set grid and block dimensions, create LOD, pass parameters to device and call kernels
	 */
	__host__ void rayTrace(glm::ivec2& texture_resolution, glm::vec3& frame_dimensions, glm::vec3& camera_forward, glm::vec3& grid_camera_pos, unsigned char* color_buffer, bool use_color_map, float max_height)
	{
		/*
		 *  Things to consider:
		 *  Branch divergence inside a warp
		 *  Maximum number threads and blocks inside a SM
		 *  Maximum number of threads per block
		 */
		dim3 gridSize, blockSize;
		cuda_setParameters << <1, 1 >> > (frame_dimensions, camera_forward, grid_camera_pos, use_color_map, max_height);
		checkCudaErrors(hipDeviceSynchronize());
		
		blockSize = dim3(1, texture_resolution.y/2);
		// ReSharper disable CppAssignedValueIsNeverUsed
		gridSize = dim3(texture_resolution.x / blockSize.x, texture_resolution.y / blockSize.y);
		cuda_rayTrace << <gridSize, blockSize >> > (color_buffer);
		checkCudaErrors(hipDeviceSynchronize());
	}

	/*
	 * Initialize variables in the device
	 */
	__host__ void initializeDeviceVariables(glm::ivec2& point_buffer_res, glm::ivec2& texture_res, float* d_gpu_pointBuffer, CudaSpace::Color* d_color_map, int LOD_levels, int stride_x, float max_height)
	{
		cuda_initializeDeviceVariables << <1, 1 >> > (point_buffer_res, texture_res, d_gpu_pointBuffer, d_color_map, LOD_levels, stride_x, max_height);
		checkCudaErrors(hipDeviceSynchronize());
	}

	/*
	 * Free memory addresses in device
	 */
	__host__ void freeDeviceVariables()
	{
		cuda_freeDeviceVariables << <1, 1 >> > ();
		checkCudaErrors(hipDeviceSynchronize());
	}
}
